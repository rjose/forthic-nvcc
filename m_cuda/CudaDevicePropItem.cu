#include "CudaDevicePropItem.h"

shared_ptr<CudaDevicePropItem> CudaDevicePropItem::New(hipDeviceProp_t value) {
    return shared_ptr<CudaDevicePropItem>(new CudaDevicePropItem(value));
}

const hipDeviceProp_t& CudaDevicePropItem::deviceProp() {
    return value;
}

string CudaDevicePropItem::StringRep() {
    return "CudaDevicePropItem";
}

string CudaDevicePropItem::AsString() {
    return "CudaDevicePropItem";
}
